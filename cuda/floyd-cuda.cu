#include <stdio.h>

#include <hip/hip_runtime.h>

#include <driver_functions.h>

#define B 16
#define RC(i, j) (i * N + j)

__global__ void floyd_kernel(int* D, int N, int k) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= N || j >= N)
    return;
  int d = D[RC(i, k)] + D[RC(k, j)];
  if (d < D[RC(i, j)]) {
    D[RC(i, j)] = d;
  }
}

void floyd_cuda(int* input, int* output, int N) {
  // compute number of blocks and threads per block
  const dim3 block_dim(B, B);
  const dim3 grid_dim((N + B - 1) / B, (N + B - 1) / B);

  int* device_data;
  hipMalloc(&device_data, N * N * sizeof(int));

  hipMemcpy(device_data, input, N * N * sizeof(int), hipMemcpyHostToDevice);

  for (int k = 0; k < N; k++) {
    floyd_kernel<<<grid_dim, block_dim>>>(device_data, N, k);
    hipDeviceSynchronize();
  }

  hipMemcpy(output, device_data, N * N * sizeof(float),
             hipMemcpyDeviceToHost);

  hipError_t errCode = hipPeekAtLastError();
  if (errCode != hipSuccess) {
    fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode,
            hipGetErrorString(errCode));
  }

  hipFree(device_data);
}

void printCudaInfo() {
  // for fun, just print out some stats on the machine

  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}
